#include "hip/hip_runtime.h"
#include "learn_atomic.cuh"
#include <iostream>


void learn_atomic() {
	int* ret_array = nullptr;
	hipError_t ret = hipMalloc(&ret_array, sizeof(int));
	if (ret != hipSuccess || ret_array == nullptr) {
		std::cerr << "Alloc device memory fail ! " << hipGetErrorString(ret) << std::endl;
		return;
	}
	learn_atomic <<<16, 512 >>> (ret_array);
	int* host_ret_array = nullptr;
	ret = hipHostMalloc(&host_ret_array, sizeof(int));
	if (ret != hipSuccess || host_ret_array == nullptr) {
		std::cerr << "Alloc host memory fail ! " << hipGetErrorString(ret) << std::endl;
		return;
	}
	ret = hipMemcpy(host_ret_array, ret_array, sizeof(int), hipMemcpyDeviceToHost);
	if (ret != hipSuccess) {
		std::cerr << "Memcpy fail ! " << hipGetErrorString(ret) << std::endl;
		return;
	}
	std::cout << "learn_atomic ret: " << host_ret_array[0];

	hipHostFree(host_ret_array);
	hipFree(ret_array);
	
}


__global__ void learn_atomic(int* out) {
	atomicAdd(out, 1);
}