#include "hip/hip_runtime.h"
#include "learn_shuffle.cuh"
#include <iostream>

void learn_shuffle() {
	int* ret_array = nullptr;
	size_t ret_size = 8 * 64;
	hipError_t ret = hipMalloc(&ret_array, ret_size * sizeof(int));
	if (ret != hipSuccess || ret_array == nullptr) {
		std::cerr << "Alloc device memory fail ! " << hipGetErrorString(ret) << std::endl;
		return;
	}
	learn_shuffle<<<8, 64 >>> (ret_array);
	int* host_ret_array = nullptr;
	ret = hipHostMalloc(&host_ret_array, ret_size * sizeof(int));
	if (ret != hipSuccess || host_ret_array == nullptr) {
		std::cerr << "Alloc host memory fail ! " << hipGetErrorString(ret) << std::endl;
		return;
	}
	ret = hipMemcpy(host_ret_array, ret_array, ret_size * sizeof(int), hipMemcpyDeviceToHost);
	if (ret != hipSuccess) {
		std::cerr << "Memcpy fail ! " << hipGetErrorString(ret) << std::endl;
		return;
	}

	std::cout << "learn_shared ret: " << std::endl;
	for (int i = 0; i < ret_size; ++i) {
		std::cout << host_ret_array[i];
		if (i + 1 < ret_size) {
			std::cout << ", ";
		}
		else {
			std::cout << std::endl;
		}
	}

	hipHostFree(host_ret_array);
	hipFree(ret_array);
}

__global__ void learn_shuffle(int* out) {
	int warp_id = threadIdx.x / warpSize;
	int lane_id = threadIdx.x % warpSize;
	int index = blockDim.x * blockIdx.x + threadIdx.x;
	out[index] = __shfl_sync(0xFFFFFFFF, index, 6, 0);  // ��lane id Ϊ6���̵߳�index�㲥��warp�ڵ������߳�
}