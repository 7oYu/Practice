#include "hip/hip_runtime.h"
#include "learn_stream.cuh"
#include <iostream>

void learn_stream() 
{
	hipStream_t stream1;  // ���ʹ���첽���ڴ濽����host memory��Ҫ�̶��ڴ棨ͨ��hipHostAlloc���䣩 
	hipStreamCreate(&stream1);
	hipStream_t stream2;
	hipStreamCreate(&stream2);
	int* data = nullptr;
	int grid_size = 32;
	int block_size = 512;
	int data_size = grid_size * block_size;
	hipMallocManaged(&data, data_size * sizeof(int));
	hipMemPrefetchAsync(data, data_size * sizeof(int), 0);
	for (size_t i = 0; i < 2; i++)
	{
		int offset = data_size / 2;
		kernal_add << <grid_size, block_size >> > (data + i * offset, offset);
	}
	hipMemPrefetchAsync(data, data_size * sizeof(int), hipCpuDeviceId);
	hipDeviceSynchronize();
	std::cout << "learn_stream ret: " << std::endl;
	for (int i = 0; i < 16; ++i) {
		std::cout << data[i];
		if (i + 1 < 16) {
			std::cout << ", ";
		}
		else {
			std::cout << std::endl;
		}
	}
	hipFree(data);
}

__global__ void kernal_add(int* out, int size) {
	int index = blockDim.x * blockIdx.x + threadIdx.x;
	if (index < size)
		out[index] += 1;
}
